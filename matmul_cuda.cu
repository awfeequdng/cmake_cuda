#include "hip/hip_runtime.h"
#include "matmul.hh"


// Matrix multiplication kernel called by MatMul()
 __global__ void matmul_kernel(Matrix A, Matrix B, Matrix C) {
    C.element[0] = A.element[0] * B.element[0];
    __syncthreads();
 }
// Matrix addition kernel called by MatMul()
 __global__ void matadd_kernel(Matrix A, Matrix B, Matrix C) {
    C.element[0] = A.element[0] + B.element[0];
    __syncthreads();
 }

 void matadd(Matrix& A, Matrix& B, Matrix& C) {
    Matrix d_A, d_B, d_C;
    auto size = A.height * A.weight * sizeof(double);
    hipMalloc(&d_A.element, size);
    hipMemcpy(d_A.element, A.element, size, hipMemcpyHostToDevice);

    size = B.height * B.weight * sizeof(double);
    hipMalloc(&d_B.element, size);
    hipMemcpy(d_B.element, B.element, size, hipMemcpyHostToDevice);

    size = C.height * C.weight * sizeof(double);
    hipMalloc(&d_C.element, size);

    matadd_kernel<<<1, 1>>>(d_A, d_B, d_C);

    hipFree(d_A.element);
    hipFree(d_B.element);
    hipFree(d_C.element);
 }

void matmul(Matrix& A, Matrix& B, Matrix& C) {
    Matrix d_A, d_B, d_C;
    auto size = A.height * A.weight * sizeof(double);
    hipMalloc(&d_A.element, size);
    hipMemcpy(d_A.element, A.element, size, hipMemcpyHostToDevice);

    size = B.height * B.weight * sizeof(double);
    hipMalloc(&d_B.element, size);
    hipMemcpy(d_B.element, B.element, size, hipMemcpyHostToDevice);

    size = C.height * C.weight * sizeof(double);
    hipMalloc(&d_C.element, size);

    matmul_kernel<<<1, 1>>>(d_A, d_B, d_C);

    hipFree(d_A.element);
    hipFree(d_B.element);
    hipFree(d_C.element);
}