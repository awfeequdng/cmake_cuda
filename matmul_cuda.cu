#include "hip/hip_runtime.h"
#include "matmul.hh"

// todo: cuda运算不能随便使用double，在编译成静态库使用时，double会编译报错，而float不会
// Matrix multiplication kernel called by MatMul()
 __global__ void matmul_kernel(Matrix A, Matrix B, Matrix C) {
    C.element[0] = A.element[0] * B.element[0];
    __syncthreads();
 }
// Matrix addition kernel called by MatMul()
 __global__ void matadd_kernel(Matrix A, Matrix B, Matrix C) {
    C.element[0] = A.element[0] + B.element[0];
    __syncthreads();
 }

 void matadd(Matrix& A, Matrix& B, Matrix& C) {
    Matrix d_A, d_B, d_C;
    auto size = A.height * A.weight * sizeof(float);
    hipMalloc(&d_A.element, size);
    hipMemcpy(d_A.element, A.element, size, hipMemcpyHostToDevice);

    size = B.height * B.weight * sizeof(float);
    hipMalloc(&d_B.element, size);
    hipMemcpy(d_B.element, B.element, size, hipMemcpyHostToDevice);

    size = C.height * C.weight * sizeof(float);
    hipMalloc(&d_C.element, size);

    matadd_kernel<<<1, 1>>>(d_A, d_B, d_C);

    hipFree(d_A.element);
    hipFree(d_B.element);
    hipFree(d_C.element);
 }

void matmul(Matrix& A, Matrix& B, Matrix& C) {
    Matrix d_A, d_B, d_C;
    auto size = A.height * A.weight * sizeof(float);
    hipMalloc(&d_A.element, size);
    hipMemcpy(d_A.element, A.element, size, hipMemcpyHostToDevice);

    size = B.height * B.weight * sizeof(float);
    hipMalloc(&d_B.element, size);
    hipMemcpy(d_B.element, B.element, size, hipMemcpyHostToDevice);

    size = C.height * C.weight * sizeof(float);
    hipMalloc(&d_C.element, size);

    matmul_kernel<<<1, 1>>>(d_A, d_B, d_C);

    hipFree(d_A.element);
    hipFree(d_B.element);
    hipFree(d_C.element);
}