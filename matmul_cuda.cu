#include "hip/hip_runtime.h"
#include "matmul.hh"


// Matrix multiplication kernel called by MatMul()
 __global__ void matmul_kernel(Matrix A, Matrix B, Matrix C) {
    C.element[0] = A.element[0] + B.element[0];
 }

void matmul(Matrix& A, Matrix& B, Matrix& C) {
    Matrix d_A, d_B, d_C;
    auto size = A.height * A.weight * sizeof(double);
    hipMalloc(&d_A.element, size);
    hipMemcpy(d_A.element, A.element, size, hipMemcpyHostToDevice);

    size = B.height * B.weight * sizeof(double);
    hipMalloc(&d_B.element, size);
    hipMemcpy(d_B.element, B.element, size, hipMemcpyHostToDevice);

    matmul_kernel<<<1, 1>>>(d_A, d_B, d_C);

    __syncthreads();

    hipFree(d_A.element);
    hipFree(d_B.element);
    hipFree(d_C.element);
}