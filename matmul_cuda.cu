#include "matmul.hh"

void matmul(Matrix& A, Matrix& B, Matrix& C) {
    Matrix d_A;
    auto size = A.height * A.weight * sizeof(double);
    hipMalloc(&d_A.element, size);
    hipMemcpy(d_A.element, A.element, size, hipMemcpyHostToDevice);

    hipFree(d_A.element);
}